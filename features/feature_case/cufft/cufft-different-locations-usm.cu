// ====------ cufft-different-locations-usm.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

size_t* work_size;
int odist;
int ostride;
int * onembed;
int idist;
int istride;
int* inembed;
int * n;
double* odata;
double2* idata;

#define HANDLE_CUFFT_ERROR( err ) (CufftHandleError( err, __FILE__, __LINE__ ))
static void CufftHandleError( hipfftResult err, const char *file, int line ) {
  if (err != HIPFFT_SUCCESS) {
    fprintf(stderr, "Cufft error in file '%s' in line %i : %s.\n",
            __FILE__, __LINE__, "error" );
  }
}

int main() {
  hipfftHandle plan1;
  hipfftResult res1 = hipfftMakePlanMany(plan1, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
  hipfftResult res2 = hipfftExecZ2D(plan1, idata, odata);

  hipfftHandle plan2;
  res1 = hipfftMakePlanMany(plan2, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
  res2 = hipfftExecZ2D(plan2, idata, odata);

  hipfftHandle plan3;
  HANDLE_CUFFT_ERROR(hipfftMakePlanMany(plan3, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size));
  HANDLE_CUFFT_ERROR(hipfftExecZ2D(plan3, idata, odata));

  hipfftHandle plan4;
  hipfftHandle plan5;
  if(hipfftMakePlanMany(plan4, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  } else if (hipfftMakePlanMany(plan5, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  }
  if (hipfftExecZ2D(plan4, idata, odata)) {
  } else if(hipfftExecZ2D(plan5, idata, odata)) {
  }

  hipfftHandle plan6;
  if(hipfftResult res = hipfftMakePlanMany(plan6, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  }
  if(hipfftResult res = hipfftExecZ2D(plan6, idata, odata)) {
  }

  hipfftHandle plan7;
  for (hipfftMakePlanMany(plan7, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);;) {
  }
  for (hipfftExecZ2D(plan7, idata, odata);;) {
  }

  hipfftHandle plan8;
  for (;hipfftMakePlanMany(plan8, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);) {
  }
  for (;hipfftExecZ2D(plan8, idata, odata);) {
  }

  hipfftHandle plan9;
  while (hipfftMakePlanMany(plan9, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size) != 0) {
  }
  while (hipfftExecZ2D(plan9, idata, odata) != 0) {
  }

  hipfftHandle plan10;
  do {
  } while (hipfftMakePlanMany(plan10, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size));
  do {
  } while (hipfftExecZ2D(plan10, idata, odata));

  hipfftHandle plan11;
  switch (int stat = hipfftMakePlanMany(plan11, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)){
  }

  switch (int stat = hipfftExecZ2D(plan11, idata, odata)){
  }
  return 0;
}

hipfftResult foo1(hipfftHandle plan) {
  return hipfftMakePlanMany(plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

hipfftResult foo2(hipfftHandle plan) {
  return hipfftExecZ2D(plan, idata, odata);
}

hipfftResult foo3(hipfftHandle plan) {
  return hipfftMakePlanMany(plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

hipfftResult foo4(hipfftHandle plan) {
  return hipfftExecZ2D(plan, idata, odata);
}

